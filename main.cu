#include "hip/hip_runtime.h"
#include "utils.cuh"
#include "nj_read.cuh"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define TPB 32

__global__ void getQMatrix(Matrix D, Vector S, Matrix Q);
__global__ void getMinQReduction(Matrix Q);
__global__ void getMinQUnroll8(Matrix Q);
__global__ void getMinQUnroll16(Matrix Q);

int main(int argc, char const *argv[])
{
    int N, iter;

    dim3 dimThread(TPB, TPB);

    nj_read read;
    Matrix h_D, d_D, d_Q;
    Vector h_S, d_S;
    size_t size_D, size_S;

    read = read_matrix("/content/drive/MyDrive/colab-data/nj-data/gen_100.ent");
    if(read.error) goto EXIT;
    
    size_D = read.size_D;
    size_S = read.size_SUM;
    h_D = read.D;
    h_S = read.SUM;
    N = read.N;

    h_D.n = N;
    h_S.size = N;
    d_D.n = N;
    d_S.size = N;
    d_Q.n = N;

    iter = N;

    hipMalloc(&d_D.elements, size_D);
    hipMalloc(&d_S.elements, size_S);
    hipMemcpy(d_D.elements, h_D.elements, size_D, hipMemcpyHostToDevice);
    hipMemcpy(d_S.elements, h_S.elements, size_S, hipMemcpyHostToDevice);

    hipMalloc(&d_Q.elements, size_D);

    // A EXECUCAO DO NJ É "LINEAR" (dependente de uma situcao anteior)
    // Somente e possivel calcular as novas distancias depois de selecionar o par
    

    while(iter == N){
        int gridX = (iter+dimThread.x -1)/(dimThread.x);
        int gridY = (iter+dimThread.y -1)/(dimThread.y);
        printf("%d x %d %d x %d\n", gridX, gridY, dimThread.x, dimThread.y);
        dim3 dimGrid(gridX, gridY);
        getQMatrix<<<dimGrid, dimThread>>>(d_D, d_S, d_Q);

        dim3 dimGrid8(dimGrid.x/8, dimGrid.y/8);
        getMinQUnroll8<<<dimGrid, dimThread>>>(d_Q);
        hipDeviceSynchronize();

        iter--;
    }

    hipFree(d_D.elements);
    hipFree(d_S.elements);
    hipFree(d_Q.elements);

    free(h_D.elements);
    free(h_S.elements);

    return 0;

    EXIT:
    hipFree(d_D.elements);
    hipFree(d_S.elements);
    hipFree(d_Q.elements);
    return 1;

    return 0;
}

__global__ void getQMatrix(Matrix D, Vector S, Matrix Q){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float value, d_rc;
    if(row == col) return;
    if(row >= D.n || col >=D.n) return;

    d_rc = d_get_matrix_position(D, row, col);
    value = (D.n-2)*d_rc - S.elements[row] - S.elements[col];

    d_set_matrix_position(Q, row, col, value);
    //printf("%.2f %f\n", value, d_rc);
}

__global__ void getMinQReduction(Matrix Q){
    // row y, col x

}

__global__ void getMinQUnroll8(Matrix Q){
    int row = threadIdx.y;
    int col = threadIdx.x;

    int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;
    int idy = blockIdx.y * blockDim.y * 8 + threadIdx.y;

    float min_8 = 0, element;

    //__shared__ float* datax = Q.elements + blockIdx.x * blockDim.x * 8;
    //__shared__ float* datay = Q.elements + blockIdx.x * blockDim.x * 8;

    float* datax = Q.elements + blockIdx.x * blockDim.x * 8;
    float* datay = Q.elements + blockIdx.y * blockDim.y * 8;

    int id_i, id_j;

    if(idx + 7 * blockDim.x < Q.n && idy + 7 * blockDim.y < Q.n){
        for (int i = 0; i < 8; i++){
            id_i = idy + i*blockDim.y;

            for(int j = 0; j < 8; j++){
                id_j = idx + i*blockDim.x;
                element = d_get_matrix_position(Q, id_i, id_j);

                if(element < min_8){
                    min_8 = element;
                }
            }
        }
    }

    __syncthreads();

    for (int stride_i = blockDim.y/2; stride_i > 0; stride_i >>= 1){

        if(row < stride_i){
            if(datay[row] > datay[row + stride_i]){
                datay[row] = datay[row + stride_i];
            }
        }
        __syncthreads();
    }

    for(int stride_j = blockDim.x/2; stride_j < 8; stride_j >>=1){
        if( col < stride_j){
            if(datax[col] > datay[col + stride_j]){
                datax[col] = datay[col + stride_j];
            }
        }
        __syncthreads();
    }

    if(datax[col] < min_8) min_8 = datax[col];
    __syncthreads();
    if(datay[row] < min_8) min_8 = datay[row];
    __syncthreads();



    if(col == 0 && row == 0){
        d_set_matrix_position(Q, blockIdx.y, blockIdx.x, min_8);
        //printf("%f\n", min_8);
    }
}

